#include "hip/hip_runtime.h"
#include <lif_cell_group_gpu.hpp>
#include <backends/gpu/kernels/stack.hpp>

using namespace arb;

// Constructor containing gid of first cell in a group and a container of all cells.
lif_cell_group_gpu::lif_cell_group_gpu(std::vector<cell_gid_type> gids, const recipe& rec):
gids_(std::move(gids))
{
    // Default to no binning of events.
    set_binning_policy(binning_kind::none, 0);

    // reserve
    cells_.reserve(gids_.size());
    // cell description variables
    tau_m_.reserve(gids_.size());
    V_th_.reserve(gids_.size());
    C_m_.reserve(gids_.size());
    E_L_.reserve(gids_.size());
    V_m_.reserve(gids_.size());
    V_reset_.reserve(gids_.size());
    t_ref_.reserve(gids_.size());

    // resize
    last_time_updated_.resize(gids_.size());

    for (auto lid: util::make_span(0, gids_.size())) {
        cells_.push_back(util::any_cast<lif_cell_description>(rec.get_cell_description(gids_[lid])));
        auto cell = cells_[lid];

        tau_m_.push_back(cell.tau_m);
        V_th_.push_back(cell.V_th);
        C_m_.push_back(cell.C_m);
        E_L_.push_back(cell.E_L);
        V_m_.push_back(cell.V_m);
        V_reset_.push_back(cell.V_reset);
        t_ref_.push_back(cell.t_ref);
    }

    // A buffer collecting the spikes produced by all cells in this group
    // during the invokation of advance(...) method
    // We assume that no neuron will spike more than 10 times
    // during one min_delay period.
    // The refractory period should prevent the neuron from spiking
    // more than this threshold.
    spike_stack = gpu::make_managed_ptr<stack_type>(gids.size() * 10);
}

cell_kind lif_cell_group_gpu::get_cell_kind() const {
    return cell_kind::lif_neuron;
}

const std::vector<spike>& lif_cell_group_gpu::spikes() const {
    return spikes_;
}

void lif_cell_group_gpu::clear_spikes() {
    spikes_.clear();
}

// TODO: implement sampler
void lif_cell_group_gpu::add_sampler(sampler_association_handle h, cell_member_predicate probe_ids,
                                    schedule sched, sampler_function fn, sampling_policy policy) {}
void lif_cell_group_gpu::remove_sampler(sampler_association_handle h) {}
void lif_cell_group_gpu::remove_all_samplers() {}

// TODO: implement binner_
void lif_cell_group_gpu::set_binning_policy(binning_kind policy, time_type bin_interval) {
}

void lif_cell_group_gpu::reset() {
    spikes_.clear();
    last_time_updated_.clear();
}

__global__
void advance_kernel (time_type tfinal,
                    unsigned num_cells,
                    double* tau_m,
                    double* V_th,
                    double* C_m,
                    double* E_L,
                    double* V_m,
                    double* V_reset,
                    double* t_ref,
                    time_type* last_time_updated,
                    pse_vector* event_lanes,
                    lif_cell_group_gpu::stack_type* spike_stack)
{
    int lid = threadIdx.x + blockIdx.x * blockDim.x;
    if (lid >= num_cells) return;

    pse_vector event_lane = event_lanes[lid];

    // Current time of last update.
    time_type t = last_time_updated[lid];
    unsigned i = 0;

    // If a neuron was in the refractory period,
    // ignore any new events that happened before t,
    // including poisson events as well.
    for (auto ev : event_lane) {
        if (ev.time >= t) break;
        ++i;
    }

    // Integrate until tfinal using the exact solution of membrane voltage differential equation.
    for (; i < event_lane.size(); i++) {
        auto ev = event_lane[i];
        if (ev.time >= tfinal) break;

        auto weight = ev.weight;
        auto time = ev.time;

        // If a neuron is in refractory period, ignore this event.
        if (time < t) continue;

        // if there are events that happened at the same time as this event, process them as well
        while (i + 1 < event_lane.size() && event_lane[i+1].time <= time) {
            weight += event_lane[i+1].weight;
            ++i;
        }

        // Let the membrane potential decay.
        V_m[lid] *= exp(-(time - t) / tau_m[lid]);
        // Add jump due to spike.
        V_m[lid] += weight/C_m[lid];
        t = time;
        // If crossing threshold occurred
        if (V_m[lid] >= V_th[lid]) {
            gpu::threshold_crossing spike;
            spike.index = lid;
            spike.time = t;
            gpu::push_back<postsynaptic_spike_event>(spike_stack->storage(), spike);

            // Advance last_time_updated.
            t += t_ref[lid];

            // Reset the voltage to the resting potential.
            V_m[lid] = E_L[lid];
        }
        // This is the last time a cell was updated.
        last_time_updated[lid] = t;
    }
}

void lif_cell_group_gpu::advance(epoch ep, time_type dt, const event_lane_subrange& event_lanes) {
    PE("lif");
    if (event_lanes.size() <= 0) return;

    unsigned block_dim = 128;
    unsigned grid_dim = (cells_.size() - 1) / block_dim + 1;

    advance_kernel<<<grid_dim, block_dim>>>(ep.tfinal,
                                            cells_.size(),
                                            tau_m_.data(),
                                            V_th_.data(),
                                            C_m_.data(),
                                            E_L_.data(),
                                            V_m_.data(),
                                            V_reset_.data(),
                                            t_ref_.data(),
                                            last_time_updated_.data(),
                                            event_lanes.data(),
                                            spike_stack.get());
    hipDeviceSynchronize();

    for (unsigned i = 0; i < spike_stack->size(); ++i) {
        gpu::threshold_crossing crossing = (*spike_stack)[i];
        spikes_.push_back(spike({gids_[crossing.index], 0}, crossing.time));
    }

    spike_stack->clear();
    PL();
}
